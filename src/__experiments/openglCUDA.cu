#include "hip/hip_runtime.h"
#ifdef __APPLE__
	#pragma clang diagnostic ignored "-Wdeprecated-declarations"
	# define __gl_h_
	# define GL_DO_NOT_WARN_IF_MULTI_GL_VERSION_HEADERS_INCLUDED
#endif

#if defined(__APPLE__) || defined(MACOSX)
	#define GLFW_INCLUDE_GLCOREARB
#endif
#include <GLFW/glfw3.h> // GLFW helper library
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
	#include <GL/wglew.h>
#endif
#include <glm/glm.hpp>
#include <glm/gtx/transform.hpp>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include "../visualization/GLShader.hpp"

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop
#include "hip/hip_runtime_api.h"
#include <hip/hip_vector_types.h>

double elapsedTime() {
	static double previous_seconds = glfwGetTime ();
	double current_seconds = glfwGetTime ();
	double elapsedTime = current_seconds - previous_seconds;
	previous_seconds = current_seconds;
	return elapsedTime;
}

void updateWindoWFPSCounter (GLFWwindow* window) {
	static double previous_seconds = glfwGetTime ();
	static int frame_count;
	double current_seconds = glfwGetTime ();
	double elapsed_seconds = current_seconds - previous_seconds;
	if (elapsed_seconds > 0.25) {
		previous_seconds = current_seconds;
		double fps = (double)frame_count / elapsed_seconds;
		char tmp[128];
		sprintf (tmp, "opengl @ fps: %.2f", fps);
		glfwSetWindowTitle (window, tmp);
		frame_count = 0;
	}
	frame_count++;
}
//-------------------------------------------------------------------





//-------------------------------------------------------------------
/*
	helper
*/
#define X 0 
#define Y 1
#define Z 2
#define D 3

/*
	memory addresses ->
	X Y Z 	X Y Z 	X Y Z
	point0  point1  point2
*/
struct Points {
	GLfloat* data;
	GLfloat get(int directionDim, int pointDim) { 
		return data[directionDim + pointDim*3]; 
	}
	void 	set(int directionDim, int pointDim, GLfloat value) { 
		data[directionDim+ pointDim*3] = value;  
	}
};

/*
	create 2D circle, point[Z] == 0 for each point, where Z==2
*/
GLfloat* createCircleOfEmotions(
	GLfloat middleX, 
	GLfloat middleY, 
	GLfloat R, 
	unsigned triangleNum) 
{
	int pointsNum = triangleNum+3;

	GLfloat radiusStep = 2*M_PI / triangleNum;

	GLfloat* data = new GLfloat[pointsNum*3];
	Points points;
	points.data = data;

	points.set(X,0, middleX);
	points.set(Y,0, middleY);
	points.set(Z,0, 0.0);

	for (int it = 0; it <= triangleNum; ++it)
	{
		GLfloat alpha = it*radiusStep;
		GLfloat deltaX = sin(alpha) * R;
		GLfloat deltaY = cos(alpha) * R;
		points.set(X, 1+it, deltaX + middleX);
		points.set(Y, 1+it, deltaY + middleY);
		points.set(Z, 1+it, 0.0);
		
		using namespace std;
		cout << "alpha == " << alpha / M_PI * 180 << endl;
		cout << points.get(X, 1+it) << endl;
		cout << points.get(Y, 1+it) << endl;
		cout << points.get(Z, 1+it) << endl << endl;

	}

	return points.data;
}

__global__
static void createCircleOfEmotionsCUDA(
	GLfloat middleX, 
	GLfloat middleY, 
	GLfloat R, 
	unsigned triangleNum, 
	GLfloat* positions)
{
	int it = threadIdx.x + blockIdx.x*blockDim.x;
	if (it < triangleNum) {
		GLfloat radiusStep = 2*M_PI / triangleNum;
		GLfloat alpha = it * radiusStep;
		GLfloat deltaX = sin(alpha) * R;
		GLfloat deltaY = cos(alpha) * R;
		positions[ it*3 		] = deltaX + middleX;
		positions[ it*3 + 1 	] = deltaY + middleY;
		positions[ it*3 + 2 	] = 0.0f;
	}
}
//-------------------------------------------------------------------





//-------------------------------------------------------------------
int main () {
	if (!glfwInit ()) {
		fprintf (stderr, "ERROR: could not start GLFW3\n");
		return 1;
	}

#if defined(__APPLE__) || defined(MACOSX)
	glfwWindowHint (GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint (GLFW_CONTEXT_VERSION_MINOR, 2);
	glfwWindowHint (GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
	glfwWindowHint (GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#endif

	glfwWindowHint (GLFW_SAMPLES, 4);
	GLFWwindow* window = 
		glfwCreateWindow (600, 600, "Hello Triangle", NULL, NULL);
	if (!window) {
		fprintf (stderr, "ERROR: could not open window with GLFW3\n");
		glfwTerminate();
		return 1;
	}
	glfwMakeContextCurrent (window);

#if defined(__APPLE__) || defined(MACOSX)
#else
	// start GLEW extension handler
	glewExperimental = GL_TRUE;
	glewInit ();
#endif

	int fbwidth, fbheight;
	glfwGetFramebufferSize(window, &fbwidth, &fbheight);
	std::cout << "fbwidth" << fbwidth << std::endl;
	std::cout << "fbheight" << fbheight << std::endl;

	// get version info
	const GLubyte* renderer = glGetString (GL_RENDERER); 	
	const GLubyte* version = glGetString (GL_VERSION); 		
	printf ("Renderer: %s\n", renderer);
	printf ("OpenGL version supported %s\n", version);

	glEnable 	(GL_DEPTH_TEST); 	
	glDepthFunc (GL_LESS); 			

	//---------------------------------------------------------------
	//Model with CUDA
	// vbo variables

	//CUDA
	cudaGLSetGLDevice(0);
	//.

	const int triangleNum = 100;
	const int vertexNum = triangleNum+3;

	GLuint vbo = 0;
	glGenBuffers (1, &vbo);
	glBindBuffer (GL_ARRAY_BUFFER, vbo);
	glBufferData (GL_ARRAY_BUFFER, 
		D * vertexNum * sizeof (float), 
		0, GL_DYNAMIC_DRAW);
	glBindBuffer (GL_ARRAY_BUFFER, 0);
	
	GLuint vao = 0;
	glGenVertexArrays (1, &vao);
	glBindVertexArray (vao);
	glEnableVertexAttribArray (0);
	glBindBuffer (GL_ARRAY_BUFFER, vbo);
	glVertexAttribPointer (0, D, GL_FLOAT, GL_FALSE, 0, NULL);
	glBindBuffer (GL_ARRAY_BUFFER, 0);
	glBindVertexArray (0);

	//CUDA
	struct hipGraphicsResource *cuda_vbo_resource = NULL;
	
	checkCudaErrors(
		hipGraphicsGLRegisterBuffer(
			&cuda_vbo_resource, vbo, 
			cudaGraphicsMapFlagsWriteDiscard));

	GLfloat *dptr;
	size_t num_bytes;
    checkCudaErrors(
    	hipGraphicsMapResources(
    		1, &cuda_vbo_resource, 0));
    checkCudaErrors(
    	hipGraphicsResourceGetMappedPointer(
    		(void **)&dptr, &num_bytes, cuda_vbo_resource));

    createCircleOfEmotionsCUDA <<<triangleNum, 1>>> (
    	0.0f, 0.0f, 0.5f, triangleNum, dptr);

    checkCudaErrors(
    	hipGraphicsUnmapResources(
    		1, &cuda_vbo_resource, 0));
	//CUDA.




	GLuint shader_programme =
		LoadShader(
			"src/visualization/shader.vert", 
			"src/visualization/shader.frag"
		);
	glm::mat4 mvp;
	GLuint MatrixID = glGetUniformLocation(shader_programme, "MVP");
	//---------------------------------------------------------------


	float angle = 0.0f;
	while (!glfwWindowShouldClose (window)) {
		//update mvp
		float deltaTime = elapsedTime();
		//angle += deltaTime * 1.0f;
		mvp = glm::rotate(angle , glm::vec3(0.0f, 0.0f, -1.0f));
		
		// wipe the drawing surface clear
		glClearColor(1,1,1,1);
		glClear (GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glUseProgram (shader_programme);
		glUniformMatrix4fv(MatrixID, 1, GL_FALSE, &mvp[0][0]);
		glBindVertexArray (vao);
		
		// draw points 0-3 from the currently 
		// bound VAO with current in-use shader
		glDrawArrays (GL_TRIANGLE_FAN, 0, vertexNum);
		
		// update other events like input handling 
		glfwPollEvents ();
		
		// put the stuff we've been drawing onto the display
		glfwSwapBuffers (window);
	}

	// close GL context and any other GLFW resources
	glfwTerminate();
}
//-------------------------------------------------------------------




