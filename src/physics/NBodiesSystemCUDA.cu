#include "hip/hip_runtime.h"
/*
	In this source file there is used macro ALIGN_INT which is define in Params.h;
*/

#include "NBodiesSystemCUDA.h"

#define SIZEF(x) (sizeof(float)*(x))
#define K 512
#define G 0.1f
#define efactor 0.1f





__device__ inline float getVal(float* data, int d, int i, int D, int N)
{
#if ALIGN_INT == INTERLEAVED_INT
	return data[i*D + d];
#endif
#if ALIGN_INT == SEPERATED_INT
	return data[d*N + i];
#endif
}

__device__ inline void setVal(float* data, int d, int i, float value, int D, int N)
{
#if ALIGN_INT == INTERLEAVED_INT
	data[i*D + d] = value;
#endif
#if ALIGN_INT == SEPERATED_INT
	data[d*N + i] = value;
#endif
}

#define GETVAL(data, d, i) getVal(data, d, i, D, N)
#define SETVAL(data, d, i, value) setVal(data, d, i, value, D, N)

__global__
static void stepKernel(
	/*Delta time*/
	float delta_t,

	/*Dimensions*/
	int N,
	int D,

	/*State*/
	float* p_prev,
	float* p_curr,
	float* v_prev,
	float* v_curr,
	float* a,
	float* m
) 
{	
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if ( i >= N ) return; 


	//--------------------------------------------------------------
	/*
		UPDATE v SPEED and p POSITION
	*/
	for (int d = 0; d < D; ++d) {
		SETVAL(v_curr, d, i, GETVAL(v_prev, d, i)  +  GETVAL(a, d, i) * delta_t);
		SETVAL(p_curr, d, i, GETVAL(p_prev, d, i)  +  (GETVAL(v_prev, d, i) + GETVAL(v_curr, d, i)) * 0.5 * delta_t);
		if (GETVAL(p_curr, d, i) > 1 || GETVAL(p_curr, d, i) < -1) 
			SETVAL(v_curr, d, i, -GETVAL(v_curr, d, i));
	}
	//--------------------------------------------------------------


	//--------------------------------------------------------------
	/*
		UPDATE a ACCELERATION
	*/
	for (int d = 0; d < D; ++d)
		SETVAL(a, d, i, 0.0f);

	for (int j = 0; j < N; ++j) {
		if ( i == j ) continue;

		/*
			delta X, delta Y, delta Z	
		*/
		position_type* r_axis = new position_type[D];
		
		position_type r_squared = 0.0f;
		for (int d = 0; d < D; ++d) {
			r_axis[d] = (GETVAL(p_curr, d, i) - GETVAL(p_curr, d, j));
			r_squared += r_axis[d] * r_axis[d];
		}

		position_type a_scalar = 
			G * GETVAL(m, 0, j) / pow(r_squared + efactor, 1.5);

		/* START DEBUG */
		// if ( a_scalar != a_scalar ) {
		// 	printf("\n");
		// 	printf("D <- %d\n", D);
		// 	printf("N <- %d\n", N);
		// 	printf("G <- %f\n", G);
		// 	printf("i <- %d\n", i);
		// 	printf("j <- %d\n", j);
		// 	printf("GETVAL(m, 0, j) <- %f\n", GETVAL(m, 0, j));
		// 	printf("GETVAL(p_curr, 0, i) <- %f\n", GETVAL(p_curr, 0, i));
		// 	printf("GETVAL(p_curr, 0, j) <- %f\n", GETVAL(p_curr, 0, j));
		// 	printf("GETVAL(p_curr, 1, i) <- %f\n", GETVAL(p_curr, 1, i));
		// 	printf("GETVAL(p_curr, 1, j) <- %f\n", GETVAL(p_curr, 1, j));
		// 	printf("GETVAL(p_curr, 2, i) <- %f\n", GETVAL(p_curr, 2, i));
		// 	printf("GETVAL(p_curr, 2, j) <- %f\n", GETVAL(p_curr, 2, j));
		// 	printf("r_squared <- %f\n", r_squared);
		// 	printf("efactor <- %f\n", efactor);
		// 	printf("\n");
		// }
		/* END__ DEBUG */
		

		for (int d = 0; d < D; ++d) {
			/*
				If both objects positions are the same
				there is division by zero; what to do then?
			 	I just set then acceleration to 0;
			*/
			if (r_axis[d])
			{
				SETVAL(a, d, i, 
					GETVAL(a, d, i) - 
						a_scalar * 
						(r_axis[d]/sqrt(r_squared)));
			}
		}

		delete [] r_axis;
	}
	//--------------------------------------------------------------
}





void NBodiesSystemCUDA::mapGLtoCUDA() {
	size_t num_bytes;
	checkCudaErrors(
    	hipGraphicsMapResources(
    		1, &cuda_vbo__p_curr, 0));
    checkCudaErrors(
    	hipGraphicsResourceGetMappedPointer(
    		(void **)&p_curr, &num_bytes, cuda_vbo__p_curr));
}
void NBodiesSystemCUDA::unmapGLtoCUDA() {
	checkCudaErrors(
    	hipGraphicsUnmapResources(
    		1, &cuda_vbo__p_curr, 0));
}

NBodiesSystemCUDA::NBodiesSystemCUDA(
	unsigned D, unsigned N,
	Params<float>& 	p_0,
	Params<float>& 	v_0,
	Params<float>& 	m_0,
	GLuint vbo__p_curr)
{
	/*
		save OpenGL VBO of buffor for p_curr;
		register GL buffer for CUDA usage;
	*/
	this->N = N;
	this->D = D;
	this->vbo__p_curr = vbo__p_curr;

	//CUDA
	cudaGLSetGLDevice(0);
	//.	

	if (vbo__p_curr != 0) {
		printf("vbo__p_curr %d\n", vbo__p_curr);

		checkCudaErrors(
			hipGraphicsGLRegisterBuffer(
				&cuda_vbo__p_curr, vbo__p_curr, 
				cudaGraphicsMapFlagsWriteDiscard));
		mapGLtoCUDA();
	}
	else {
		/*
			allocate memory on device
		*/
		checkCudaErrors(hipMalloc((void **)&p_curr, SIZEF(D*N)));
	}

	/*
		allocate memory on device
	*/
	checkCudaErrors(hipMalloc((void **)&p_prev, SIZEF(D*N)));
	checkCudaErrors(hipMalloc((void **)&v_prev, SIZEF(D*N)));
	checkCudaErrors(hipMalloc((void **)&v_curr, SIZEF(D*N)));
	checkCudaErrors(hipMalloc((void **)&a, 	 SIZEF(D*N)));
	checkCudaErrors(hipMalloc((void **)&m, 	 SIZEF(N  )));

	/*
		copy memory from host to device
	*/
	checkCudaErrors(hipMemcpy(p_prev, p_0.data, SIZEF(D*N), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(p_curr, p_0.data, SIZEF(D*N), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(v_prev, v_0.data, SIZEF(D*N), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(v_curr, v_0.data, SIZEF(D*N), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(m,      m_0.data, SIZEF(N),   hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(a, 0, SIZEF(D*N)));
	checkCudaErrors(hipDeviceSynchronize());

	if (vbo__p_curr != 0)
		unmapGLtoCUDA();
}

void NBodiesSystemCUDA::step( float delta_t ) {
	/**
		CUDA OpenGL interoperability; 
	**/
	if (vbo__p_curr != 0) 
		mapGLtoCUDA();

	/*
		Set curr to prev;

		p_prev = p_curr;
		v_prev = v_curr;
	*/
	checkCudaErrors(hipMemcpy(p_prev, p_curr, SIZEF(D*N), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(v_prev, v_curr, SIZEF(D*N), hipMemcpyDeviceToDevice));

	/*
		1) UPDATE v SPEED and p POSITION

		For each body update v SPEED and p POSITION;
		N*D threads;


		2)UPDATE a ACCELERATION for each body;

		we have N bodies;
		square matrix;
		a_from_i_j
	*/
	stepKernel <<<(N+K-1)/K, K>>> (
		delta_t,
		N, D,
		p_prev, p_curr,
		v_prev, v_curr,
		a, m
	);

	checkCudaErrors(hipDeviceSynchronize());

	#ifdef DEBUG_PRINT_POINTS_POSITIONS
	printState();
	#endif

	/**
		CUDA OpenGL interoperability; 
	**/
	if (vbo__p_curr != 0) 
		unmapGLtoCUDA();
}

#ifdef DEBUG_PRINT_POINTS_POSITIONS
void NBodiesSystemCUDA::printState() {
	Params<float> 	p__(D,N, ALIGN);
	Params<float>	v__(D,N, ALIGN);
	Params<float> 	a__(D,N, ALIGN);

	checkCudaErrors(hipMemcpy(p__.data, p_curr, SIZEF(D*N), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(v__.data, v_curr, SIZEF(D*N), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(a__.data, a,		 SIZEF(D*N), hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());

	for (int i = 0; i < N; ++i) {
		printf("CUDABody(%d): \n", i);
		
		printf("\t p <- (");
		for (int d = 0; d < D; ++d)
			printf("%f ", p__.getVal(d, i));
		printf(")\n");

		printf("\t v <- (");
		for (int d = 0; d < D; ++d)
			printf("%f ", v__.getVal(d, i));
		printf(")\n");

		printf("\t a <- (");
		for (int d = 0; d < D; ++d)
			printf("%f ", a__.getVal(d, i));
		printf(")\n");
	}
	printf("\n");
}
#endif
